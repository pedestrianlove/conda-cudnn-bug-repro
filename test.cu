
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <stdio.h>

int main() {
  cudnnHandle_t handle;
  cudnnStatus_t status = cudnnCreate(&handle);

  if (status != CUDNN_STATUS_SUCCESS) {
    printf("CUDNN initialization failed: %s\n", cudnnGetErrorString(status));
    return 1;
  }

  printf("CUDNN initialized successfully.\n");

  // Clean up
  cudnnDestroy(handle);
  return 0;
}
